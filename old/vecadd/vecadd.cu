#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <r.h>

__host__ void vecadd_cuda(float* a, float* b, float* c, int n) {
    int size = n * sizeof(float);
    float* aD;
    float* bD;
    float* cD;

    // allocate memory for vector and copy to device
    hipMalloc((void**)&aD, size);
    hipMemcpy(aD, a, size, hipMemcpyHostToDevice);

    hipMalloc((void**)&bD, size);
    hipMemcpy(bD, b, size, hipMemcpyHostToDevice);

    hipMalloc((void**)&cD, size);
    hipMemcpy(cD, c, size, hipMemcpyHostToDevice);

    // setup cuda kernel
    int num_blocks = ceil(n / BLOCK_WIDTH);
    dim3 dimGrid(num_blocks, 1, 1);
    dim3 dimBlock(BLOCK_WIDTH, 1, 1);

    // start cuda kernel
    vec_add_kernel << < dimGrid, dimBlock >> > (aD, bD, cD, n);

	// waits for the kernel to finish and returns errors
	hipDeviceSynchronize();

	// copy c from device
	hipMemcpy(c, cD, size, hipMemcpyDeviceToHost);

	// free device
	hipFree(aD);
	hipFree(bD);
	hipFree(cD);
}

__global__ void vec_add_kernel(float* a, float* b, float* c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}
